#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include <string>
#include <iostream>
#include <fstream>
#include <sstream>
#include <stdint.h>
#include <vector>
#define long int64_t
// TODO: Fix speed calculation when these are different numbers.
#define INPUT_BLOCK_SIZE (2 << 20)
#define WORK_UNIT_SIZE (2 << 20)
#define CHECK_GPU_ERR(code) gpuAssert((code), __FILE__, __LINE__)

#define TREE_ATTEMPTS 12

#define RANDOM_MASK (1ULL << 48) - 1
#define setSeed(rand, val) ((rand) = ((val) ^ 0x5DEECE66DLL) & ((1LL << 48) - 1))
#define advance(rand, multiplier, addend) ((rand) = ((rand) * (multiplier) + (addend)) & (RANDOM_MASK))
#define advance_1(rand) advance(rand, 0x5DEECE66DLL, 0xBLL)
#define advance_16(rand) advance(rand, 0x6DC260740241LL, 0xD0352014D90LL)
#define advance_3760(rand) advance(rand, 0x8C35C76B80C1LL, 0xD7F102F24F30LL)

struct range {
	int l, h;
};
struct tree {
	range x, y, h;
};
struct chunk {
	long x, y;
	std::vector<tree> trees;
};

__host__ __device__ int next(long *rand, int bits) {
	*rand = (*rand * 0x5DEECE66DLL + 0xBLL) & ((1LL << 48) - 1);
	return (int)(*rand >> (48 - bits));
}
__host__ __device__ long nextLong(long *rand) {
	return ((long)next(rand, 32) << 32) + next(rand, 32);
}
__host__ __device__ int nextIntBound(long *rand, int bound) {
	return (int)((bound * (long)next(rand, 31)) >> 31);
}

inline void gpuAssert(hipError_t code, const char* file, int line) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s (code %d) %s %d\n", hipGetErrorString(code), code, file, line);
        exit(code);
    }
}

// TODO: Check multiple chunks
__global__ void process(long* seeds, long offset) {
	std::vector<chunk> chunks = {
		{3, 4, {
			{{ 4,  4}, { 0,  0}, { 6,  6}},
			{{13, 13}, {14, 14}, { 4,  4}},
			{{13, 13}, { 3,  3}, { 5,  5}},
			{{12, 12}, {11, 11}, { 6,  6}},
			{{10, 10}, { 2,  2}, { 4,  4}},
		}}
	};
	long index = offset + blockIdx.x * blockDim.x + threadIdx.x;
	long seed = seeds[index];
	long rand;
	for(int c = 0; c < chunks.size(); c++) {
		setSeed(rand, seed);
		long chunkSeed = chunks[c].x * (nextLong(&rand) / 2LL * 2LL + 1LL) + chunks[c].y * (nextLong(&rand) / 2LL * 2LL + 1LL) ^ seed;
		setSeed(rand, chunkSeed);
		advance_3760(rand);
		int found = 0;
		for (int attempt = 0; attempt < TREE_ATTEMPTS; attempt++) {
			int treeX = nextIntBound(&rand, 16);
			int treeZ = nextIntBound(&rand, 16);
			int treeH = nextIntBound(&rand, 3) + 4;
			for (int tree = 0; tree < chunks[c].trees.size(); tree++) {
				bool valid = true;
				valid &= treeX >= chunks[c].trees[tree].x.l;
				valid &= treeX <= chunks[c].trees[tree].x.h;
				valid &= treeZ >= chunks[c].trees[tree].y.l;
				valid &= treeZ <= chunks[c].trees[tree].y.h;
				valid &= treeH >= chunks[c].trees[tree].h.l;
				valid &= treeH <= chunks[c].trees[tree].h.h;
				if (valid) {
					advance_16(rand);
					found++;
				};
			}
		}
		if (found == chunks[c].trees.size()) {
			printf("Seed: %lld.\n", seed);
		}
	}
}
int main(void) {
	// Allocate RAM for input
	long *input = (long *)malloc(sizeof(long) * INPUT_BLOCK_SIZE);
	// Open File
	std::ifstream ifs ("input.txt");
	if (ifs.fail()) {
		std::cout << "ERROR::IFSTREAM::FAIL" << std::endl;
		return -1;
	}
	// Allocate VRAM for input
	long *seeds;
	CHECK_GPU_ERR(hipMallocManaged((long **)&seeds, sizeof(long) * INPUT_BLOCK_SIZE));
	// Load Input Block
	// TODO: Fix issue where the last iteration will recheck seeds from the previous
	// iteration if the number of inputs is not evenly divisible by WORK_UNIT_SIZE
	// Currenmtly "fixed" by setting all remaining seeds to 0.
	// clock_t startTime = clock();
	// clock_t lastIteration = clock();
	// hipEvent_t readStart, readStop, processStart, processStop, memcpyStart, memcpyStop;
	// hipEventCreate(&readStart);
	// hipEventCreate(&readStop);
	// hipEventCreate(&processStart);
	// hipEventCreate(&processStop);
	// hipEventCreate(&memcpyStart);
	// hipEventCreate(&memcpyStop);
	std::string line;
	// float time;
	while(std::getline(ifs, line)) {
		// hipEventRecord(readStart, 0);
		for (long i = 0; i < INPUT_BLOCK_SIZE; i++) {
			if (ifs >> line) {
				long val = std::atoll(line.c_str());
				input[i] = val;
			} else {
				input[i] = 0;
			}
		}
		// hipEventRecord(readStop, 0);
		// hipEventSynchronize(readStop);
		// hipEventElapsedTime(&time, readStart, readStop);
		// printf("Read: %f\n", time);
		// Copy to VRAM
		// hipEventRecord(memcpyStart, 0);
		CHECK_GPU_ERR(hipMemcpy(seeds, input, sizeof(long) * INPUT_BLOCK_SIZE, hipMemcpyHostToDevice));
		// hipEventRecord(memcpyStop, 0);
		// hipEventSynchronize(memcpyStop);
		// hipEventElapsedTime(&time, memcpyStart, memcpyStop);
		// printf("Memcpy: %f\n", time);
		for(int offset = 0; offset < INPUT_BLOCK_SIZE; offset += WORK_UNIT_SIZE) {
			// hipEventRecord(processStart, 0);
			// Process input
			process<<<WORK_UNIT_SIZE / 256, 256>>>(seeds, offset);
			// hipDeviceSynchronize();
			// hipEventRecord(processStop, 0);
			// hipEventSynchronize(processStop);
			// hipEventElapsedTime(&time, processStart, processStop);
			// printf("Process: %f\n", time);

			// double iterationTime = (double)(clock() - lastIteration) / CLOCKS_PER_SEC;
			// double timeElapsed = (double)(clock() - startTime) / CLOCKS_PER_SEC;
			// lastIteration = clock();
			// double speed = (double)WORK_UNIT_SIZE / (double)iterationTime / 1000000.0;
			// printf("Uptime: %.1fs. Speed: %.2fm/s.\n", timeElapsed, speed);
		}
	}
	// hipEventDestroy(readStart);
	// hipEventDestroy(readStop);
	// hipEventDestroy(processStart);
	// hipEventDestroy(processStop);
	hipFree(seeds);
	ifs.close();
	return 0;
}