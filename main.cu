
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <string>
#include <iostream>
#include <fstream>
#include <sstream>
#include <stdint.h>
#include <vector>
#define long int64_t
// TODO: Fix speed calculation when these are different numbers.
#define INPUT_BLOCK_SIZE (2 << 20)
#define WORK_UNIT_SIZE (2 << 20)
#define CHECK_GPU_ERR(code) gpuAssert((code), __FILE__, __LINE__)

#ifndef CHUNK_X
#define CHUNK_X 0
#endif
#ifndef CHUNK_Y
#define CHUNK_Y 0
#endif

#define TREE_ATTEMPTS 12

#define RANDOM_MASK (1ULL << 48) - 1
#define setSeed(rand, val) ((rand) = ((val) ^ 0x5DEECE66DLL) & ((1LL << 48) - 1))
#define advance(rand, multiplier, addend) ((rand) = ((rand) * (multiplier) + (addend)) & (RANDOM_MASK))
#define advance_1(rand) advance(rand, 0x5DEECE66DLL, 0xBLL)
#define advance_16(rand) advance(rand, 0x6DC260740241LL, 0xD0352014D90LL)
#define advance_3760(rand) advance(rand, 0x8C35C76B80C1LL, 0xD7F102F24F30LL)

__host__ __device__ int next(long *rand, int bits) {
	*rand = (*rand * 0x5DEECE66DLL + 0xBLL) & ((1LL << 48) - 1);
	return (int)(*rand >> (48 - bits));
}
__host__ __device__ long nextLong(long *rand) {
	return ((long)next(rand, 32) << 32) + next(rand, 32);
}
__host__ __device__ int nextIntBound(long *rand, int bound) {
	return (int)((bound * (long)next(rand, 31)) >> 31);
}

inline void gpuAssert(hipError_t code, const char* file, int line) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s (code %d) %s %d\n", hipGetErrorString(code), code, file, line);
        exit(code);
    }
}

struct Chunk {
	int x, y;
	__host__ __device__ bool operator==(const Chunk &rhs) {
		return this->x == rhs.x && this->y == rhs.y;
	}
};

struct Tree {
	int x, y, h;
	__host__ __device__ bool operator==(const Tree &rhs) {
		return this->x == rhs.x && this->y == rhs.y && this->h == rhs.h;
	}
	__host__ __device__ bool operator>=(const Tree &rhs) {
		return this->x >= rhs.x && this->y >= rhs.y && this->h >= rhs.h;
	}
	__host__ __device__ bool operator<=(const Tree &rhs) {
		return this->x <= rhs.x && this->y <= rhs.y && this->h <= rhs.h;
	}
};

__constant__ Chunk chunks[] = {{3, 4}};

__device__ bool checkTree(int chunkIndex, Tree tree) {
	switch(chunkIndex) {
		case 0:
			if (tree == Tree{ 4,  0, 6}) return true;
			else if (tree == Tree{13, 14, 4}) return true;
			else if (tree == Tree{13,  3, 5}) return true;
			else if (tree == Tree{12, 11, 6}) return true;
			else if (tree == Tree{10,  2, 4}) return true;
		break;
		default:
		break;
	}
	return false;
}

// TODO: Allow trees to have a range of possible coordinates.
__global__ void process(long* seeds, long offset) {
	long index = offset + blockIdx.x * blockDim.x + threadIdx.x;
	long seed = seeds[index];
	long rand;
	for(int c = 0; c < sizeof(chunks) / sizeof(Chunk); c++) {
		setSeed(rand, seed);
		long chunkSeed = (chunks[c].x + CHUNK_X) * (nextLong(&rand) / 2LL * 2LL + 1LL) + (chunks[c].y + CHUNK_Y) * (nextLong(&rand) / 2LL * 2LL + 1LL) ^ seed;
		setSeed(rand, chunkSeed);
		advance_3760(rand);
		int found = 0;
		for (int attempt = 0; attempt < TREE_ATTEMPTS; attempt++) {
			Tree tree = {nextIntBound(&rand, 16), nextIntBound(&rand, 16), nextIntBound(&rand, 3) + 4};
			if (checkTree(c, tree)) {
				advance_16(rand);
				found++;
			};
		}
		if (found == 5) {
			printf("Seed: %lld.\n", seed);
		}
	}
}
//TODO: Fix timing. Using a cudaEvent_t multiple times in a loop doesn't work properly(?). Also figure out proper synchronization calls.
int main(void) {
	// Allocate RAM for input
	long *input = (long *)malloc(sizeof(long) * INPUT_BLOCK_SIZE);
	// Open File
	std::ifstream ifs ("input.txt");
	if (ifs.fail()) {
		std::cout << "ERROR::IFSTREAM::FAIL" << std::endl;
		return -1;
	}
	// Allocate VRAM for input
	long *seeds;
	CHECK_GPU_ERR(hipMallocManaged((long **)&seeds, sizeof(long) * INPUT_BLOCK_SIZE));
	// Load Input Block
	// TODO: Fix issue where the last iteration will recheck seeds from the previous
	// iteration if the number of inputs is not evenly divisible by WORK_UNIT_SIZE
	// Currently "fixed" by setting all remaining seeds to 0.
	// clock_t startTime = clock();
	// clock_t lastIteration = clock();
	hipEvent_t readStart, readStop, processStart, processStop, memcpyStart, memcpyStop;
	hipEventCreate(&readStart);
	hipEventCreate(&readStop);
	hipEventCreate(&processStart);
	hipEventCreate(&processStop);
	hipEventCreate(&memcpyStart);
	hipEventCreate(&memcpyStop);
	std::string line;
	float time;
	while(std::getline(ifs, line)) {
		hipEventRecord(readStart, 0);
		for (long i = 0; i < INPUT_BLOCK_SIZE; i++) {
			if (ifs >> line) {
				long val = std::atoll(line.c_str());
				input[i] = val;
			} else {
				input[i] = 0;
			}
		}
		hipEventRecord(readStop, 0);
		hipEventSynchronize(readStop);
		hipEventElapsedTime(&time, readStart, readStop);
		printf("Read: %f\n", time);
		// Copy to VRAM
		hipEventRecord(memcpyStart, 0);
		CHECK_GPU_ERR(hipMemcpy(seeds, input, sizeof(long) * INPUT_BLOCK_SIZE, hipMemcpyHostToDevice));
		hipEventRecord(memcpyStop, 0);
		hipEventSynchronize(memcpyStop);
		hipEventElapsedTime(&time, memcpyStart, memcpyStop);
		printf("Memcpy: %f\n", time);
		for(int offset = 0; offset < INPUT_BLOCK_SIZE; offset += WORK_UNIT_SIZE) {
			hipEventRecord(processStart, 0);
			// Process input
			process<<<WORK_UNIT_SIZE / 256, 256>>>(seeds, offset);
			hipDeviceSynchronize();
			hipEventRecord(processStop, 0);
			hipEventSynchronize(processStop);
			hipEventElapsedTime(&time, processStart, processStop);
			printf("Process: %f\n", time);

			// double iterationTime = (double)(clock() - lastIteration) / CLOCKS_PER_SEC;
			// double timeElapsed = (double)(clock() - startTime) / CLOCKS_PER_SEC;
			// lastIteration = clock();
			// double speed = (double)WORK_UNIT_SIZE / (double)iterationTime / 1000000.0;
			// printf("Uptime: %.1fs. Speed: %.2fm/s.\n", timeElapsed, speed);
		}
	}
	hipEventDestroy(readStart);
	hipEventDestroy(readStop);
	hipEventDestroy(processStart);
	hipEventDestroy(processStop);
	hipFree(seeds);
	ifs.close();
	return 0;
}